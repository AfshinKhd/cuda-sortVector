#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * sort_vec.cu - Sorting a vector using CUDA C
 *
 * Created by Afshin Khodaveisi Afshin.khodaveisi@studio.unibo.it
 *
 * ---------------------------------------------------------------------------
 *
 * Implementing vector sorting algorithms using Cuda C
 *
 * Implemented algorithms including:
 * Merge Sort - Sequential Implementation (1 block , 1 Thread)
 * Extended Merge Sort - Parallel Implementation (Multi Threads)
 ****************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define N (4 * 4)
#define BLKDIM (2)
#define BLKSIZE (2)
#define SHOWITEMS (16)



void printArray(int arr[], int n) {
	for(int unused_i=0 ; unused_i< SHOWITEMS ; unused_i++){
		printf("%d ", *(arr+unused_i));
		if (unused_i % 10 == 0 && unused_i != 0)
			printf("\n");
	}
	printf("...\n");

}

void showTime(double start, double end , char msg[])
{
	double elapsed = end - start ;
	printf("Elapsed time in %s: %.6f seconds\n",msg, elapsed);
}

double get_time(){
	clock_t	time = clock();
	return (double)(time / CLOCKS_PER_SEC);
}


void initilizeArray(int* arr) {
	for (int unused_i = 0; unused_i < N; unused_i++)
		*(arr + unused_i) = rand();
}

__device__ void _merge(int* arr, int* left, int leftSize, int* right, int rightSize) {
	int i = 0, j = 0, k = 0;
	//printf("_merge : lef[0]:%d , arr[0]:%d , right[0]:%d , arr[1]:%d\n", left[0], arr[0], right[0], arr[1]);
	while (i < leftSize && j < rightSize) {
		if (left[i] <= right[j]) {
			//printf("$$$$$$$$$\n");
			arr[k++] = left[i++];
			//printf("k:%d\n", k);
		}
		else {
			//printf("%%%%%%%%%%%\n");
			arr[k++] = right[j++];
		}
	}

	while (i < leftSize) {
		arr[k++] = left[i++];
	}

	while (j < rightSize) {
		arr[k++] = right[j++];
	}
}

__device__ void _merge_shared(int* arr, int* left, int leftStart, int leftSize, int* right, int rightStart, int rightSize) {
	int i = leftSize, j = rightStart, k = 0;
	printf("_merge : lef[0]:%d , left[1]:%d, arr[0]:%d , right[0]:%d ,right[1]:%d, arr[1]:%d\n", left[0],left[1], arr[0], right[0],right[1], arr[1]);
	while (i < leftSize && j < rightSize) {
		if (left[i] <= right[j]) {
			//printf("$$$$$$$$$\n");
			arr[k++] = left[i++];
			//printf("k:%d\n", k);
		}
		else {
			//printf("%%%%%%%%%%%\n");
			arr[k++] = right[j++];
		}
	}

	while (i < leftSize) {
		arr[k++] = left[i++];
	}

	while (j < rightSize) {
		arr[k++] = right[j++];
	}
}

__global__ void mergeSort(int* arr, int n) {
	int currentSize, leftStart;

	for (currentSize = 1; currentSize < n; currentSize *= 2) {
		for (leftStart = 0; leftStart < n - 1; leftStart += 2 * currentSize) {
			int mid = leftStart + currentSize - 1;
			int rightEnd = (leftStart + 2 * currentSize - 1 < n - 1) ? leftStart + 2 * currentSize - 1 : n - 1;
			int leftSize = mid - leftStart + 1;
			int rightSize = rightEnd - mid;

			int* left = (int*)malloc(leftSize * sizeof(int));
			int* right = (int*)malloc(rightSize * sizeof(int));
			//int left[leftSize], right[rightSize];

			// Copy data from original array to temporary left and right arrays
			for (int i = 0; i < leftSize; i++) {
				left[i] = *(arr + leftStart + i);
			}
			for (int i = 0; i < rightSize; i++) {
				right[i] = *(arr + mid + 1 + i);
			}

			_merge(arr + leftStart, left, leftSize, right, rightSize);
	
			free(left);
			free(right);
		}
	}
}

__global__ void mergeSortWithThreads(int* arr, int n) {
	int currentSize, leftStart;
	int idx = threadIdx.x;
	int partSize = blockDim.x;
	int nPart = (n + blockDim.x - 1) / blockDim.x;

	for (currentSize = 1; currentSize < n; currentSize *= 2) {
		for(int p = 0 ; p < nPart ; p++){
			int leftStart = (idx + (p * partSize)) * 2 * currentSize;
			//if (currentSize > 512 * 1   ) printf("leftstart: %d , p: %d , n: %d , partSize: %d , nPart: %d ,idx: %d\n", leftStart, p, n , partSize, nPart,idx);
			if (leftStart + currentSize > n) break;
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("SECOND: leftstart: %d , p: %d , n: %d , partSize: %d , nPart: %d ,idx: %d ,currentsize :%d\n", leftStart, p, n, partSize, nPart, idx,currentSize);
				int mid = leftStart + currentSize - 1;
				int rightEnd = (leftStart + 2 * currentSize - 1 < n - 1) ? leftStart + 2 * currentSize - 1 : n - 1;
				int leftSize = mid - leftStart + 1;
				int rightSize = rightEnd - mid;
				//if (currentSize > 512 * 1 && leftStart > 2095103) printf("leftsize: %d , rightsize: %d", leftSize, rightSize);
				//printf("index:%d , mid:%d , rightEnd:%d , leftsize:%d , rightsize:%d\n", index, mid, rightEnd, leftSize, rightSize);
				int* left = (int*)malloc(leftSize * sizeof(int));
				int* right = (int*)malloc(rightSize * sizeof(int));
				//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here1\n");
				// Copy data from original array to temporary left and right arrays
				for (int i = 0; i < leftSize; i++) {
					//if (currentSize > 512 * 1 && leftStart > 2095103) printf("########\n");
					left[i] = *(arr + leftStart + i);
				}
				//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here2\n");
				for (int i = 0; i < rightSize; i++) {
					right[i] = *(arr + mid + 1 + i);
				}
				//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here3\n");
				//printf("before : arr0:%d , arr1:%d , arr2:%d , arr3:%d\n", arr[0], arr[1], arr[2], arr[3]);
				_merge(arr + leftStart, left, leftSize, right, rightSize);
				//printf("after : arr0:%d , arr1:%d , arr2:%d , arr3:%d\n", arr[0], arr[1], arr[2], arr[3]);
				free(left);
				free(right);
				
				__syncthreads();
			
		}
	}
}

__global__ void mergeSortWithThreadsAndBlocks_limited(int* arr, int n) {
	int currentSize, leftStart;
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	int partSize = BLKSIZE * blockDim.x;
	//Warning : partsize * 2 ?????
	int nPart = (n + partSize - 1) / partSize;

	for (currentSize = 1; currentSize < n; currentSize *= 2) {
		for (int p = 0; p < nPart; p++) {
			int leftStart = (idx + (p * partSize)) * 2 * currentSize;
			//if (currentSize > 512 * 1   ) printf("leftstart: %d , p: %d , n: %d , partSize: %d , nPart: %d ,idx: %d\n", leftStart, p, n , partSize, nPart,idx);
			if (leftStart + currentSize > n) break;
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("SECOND: leftstart: %d , p: %d , n: %d , partSize: %d , nPart: %d ,idx: %d ,currentsize :%d\n", leftStart, p, n, partSize, nPart, idx,currentSize);
			int mid = leftStart + currentSize - 1;
			int rightEnd = (leftStart + 2 * currentSize - 1 < n - 1) ? leftStart + 2 * currentSize - 1 : n - 1;
			int leftSize = mid - leftStart + 1;
			int rightSize = rightEnd - mid;
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("leftsize: %d , rightsize: %d", leftSize, rightSize);
			//printf("index:%d , mid:%d , rightEnd:%d , leftsize:%d , rightsize:%d\n", index, mid, rightEnd, leftSize, rightSize);
			int* left = (int*)malloc(leftSize * sizeof(int));
			int* right = (int*)malloc(rightSize * sizeof(int));
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here1\n");
			// Copy data from original array to temporary left and right arrays
			for (int i = 0; i < leftSize; i++) {
				//if (currentSize > 512 * 1 && leftStart > 2095103) printf("########\n");
				left[i] = *(arr + leftStart + i);
			}
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here2\n");
			for (int i = 0; i < rightSize; i++) {
				right[i] = *(arr + mid + 1 + i);
			}
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here3\n");
			//printf("before : arr0:%d , arr1:%d , arr2:%d , arr3:%d\n", arr[0], arr[1], arr[2], arr[3]);
			_merge(arr + leftStart, left, leftSize, right, rightSize);
			//printf("after : arr0:%d , arr1:%d , arr2:%d , arr3:%d\n", arr[0], arr[1], arr[2], arr[3]);
			free(left);
			free(right);

			__syncthreads();

		}
	}
}

__global__ void mergeSortWithThreadsAndBlocks_limited_shared(int* arr, int n) {
	int currentSize, leftStart;
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	int partSize = BLKSIZE * blockDim.x;
	int nPart = (n + partSize - 1) / (partSize * 2);
	int blockSize = blockDim.x;
	__shared__ int* left;
	__shared__ int* right;
	//extern __shared__ int sharedArray[];

	for (currentSize = 1; currentSize < n; currentSize *= 2) {
		const int sharedSize_side =  (blockSize* currentSize * 2 < n) ? blockSize * currentSize : n/2 ;
		printf("sharedsize_side: %d , currentsize: %d \n", sharedSize_side , currentSize);
		left = (int*)malloc(sharedSize_side * sizeof(int));
		right = (int*)malloc(sharedSize_side * sizeof(int));
		//int* left = &sharedArray[sharedSize_side];
		//int* right = &sharedArray[sharedSize_side];

		for (int p = 0; p < nPart; p++) {
			int leftStart = (idx + (p * partSize)) * 2 * currentSize;
			
			if (leftStart + currentSize > n) break;
			printf("leftstart: %d , p: %d , n: %d , partSize: %d , nPart: %d ,idx: %d\n", leftStart, p, n, partSize, nPart, idx);
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("SECOND: leftstart: %d , p: %d , n: %d , partSize: %d , nPart: %d ,idx: %d ,currentsize :%d\n", leftStart, p, n, partSize, nPart, idx,currentSize);
			int mid = leftStart + currentSize - 1;
			int rightEnd = (leftStart + 2 * currentSize - 1 < n - 1) ? leftStart + 2 * currentSize - 1 : n - 1;
			int leftSize = mid - leftStart + 1;
			int rightSize = rightEnd - mid;
			int rightStart = mid + 1;
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("leftsize: %d , rightsize: %d", leftSize, rightSize);
			//printf("index:%d , mid:%d , rightEnd:%d , leftsize:%d , rightsize:%d\n", index, mid, rightEnd, leftSize, rightSize);
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here1\n");
			// Copy data from original array to temporary left and right arrays
			int _unused_idx = 0;
			for (int i = 0; i < leftSize; i++) {
				//if (currentSize > 512 * 1 && leftStart > 2095103) printf("########\n");
				_unused_idx = i % sharedSize_side;
				left[leftStart % sharedSize_side] = *(arr + leftStart + i);
			}
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here2\n");
			for (int i = 0; i < rightSize; i++) {
				_unused_idx = i % sharedSize_side;
				right[i % sharedSize_side] = *(arr + rightStart + i);
			}

			__syncthreads();
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here3\n");
			printf("before : arr0:%d , arr1:%d , arr2:%d , arr3:%d\n", arr[0], arr[1], arr[2], arr[3]);
			printf("leftStart: %d , rigfhtStart: %d , sharedSize: %d\n", leftStart, rightStart, sharedSize_side);
			_merge_shared(arr + leftStart, left , (leftStart% sharedSize_side) , leftSize + leftStart, right,(rightStart% sharedSize_side), rightSize + rightStart);
			printf("after : arr0:%d , arr1:%d , arr2:%d , arr3:%d\n", arr[0], arr[1], arr[2], arr[3]);
			__syncthreads();
	
		}

		//free(left);
		//free(right);
		
	}
}

__global__ void mergeSortWithBlocksAndThreads(int* arr, int n) {
	int currentSize = 1, leftStart;
	int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	int partSize = blockDim.x;
	int nPart = (n + blockDim.x - 1) / blockDim.x;

	while (nPart > 0) {
		
			int leftStart = idx * 2 * currentSize;
			//printf("leftstart: %d , n: %d , partSize: %d , nPart: %d ,idx: %d , currentsize:%d\n", leftStart, n , partSize, nPart,idx, currentSize);
			if (leftStart + currentSize > n) break;
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("SECOND: leftstart: %d , p: %d , n: %d , partSize: %d , nPart: %d ,idx: %d ,currentsize :%d\n", leftStart, p, n, partSize, nPart, idx,currentSize);
			int mid = leftStart + currentSize - 1;
			int rightEnd = (leftStart + 2 * currentSize - 1 < n - 1) ? leftStart + 2 * currentSize - 1 : n - 1;
			int leftSize = mid - leftStart + 1;
			int rightSize = rightEnd - mid;
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("leftsize: %d , rightsize: %d", leftSize, rightSize);
			//printf("index:%d , mid:%d , rightEnd:%d , leftsize:%d , rightsize:%d\n", index, mid, rightEnd, leftSize, rightSize);
			int* left = (int*)malloc(leftSize * sizeof(int));
			int* right = (int*)malloc(rightSize * sizeof(int));
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here1\n");
			// Copy data from original array to temporary left and right arrays
			for (int i = 0; i < leftSize; i++) {
				//if (currentSize > 512 * 1 && leftStart > 2095103) printf("########\n");
				left[i] = *(arr + leftStart + i);
			}
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here2\n");
			for (int i = 0; i < rightSize; i++) {
				right[i] = *(arr + mid + 1 + i);
			}
			//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here3\n");
			//printf("before : arr0:%d , arr1:%d , arr2:%d , arr3:%d\n", arr[0], arr[1], arr[2], arr[3]);
			_merge(arr + leftStart, left, leftSize, right, rightSize);
			//printf("after : arr0:%d , arr1:%d , arr2:%d , arr3:%d\n", arr[0], arr[1], arr[2], arr[3]);
			free(left);
			free(right);

			__syncthreads();

			nPart /= 2;
			currentSize *= 2;
		
	}
}

__global__ void mergeSortWithBlocks(int* arr, int n) {
	int currentSize, leftStart;
	int idx = blockIdx.x;
	//int partSize = 16;
	int nPart = n/2 ; 

	
			while (nPart > 0) {
				currentSize = n / (nPart * 2);
				int leftStart = idx * 2 * currentSize;
				if (leftStart + currentSize > n) break;
				//printf("npart: %d , idx: %d , currentsize: %d , leftStart:%d\n", nPart, idx, currentSize, leftStart);
				int mid = leftStart + currentSize - 1;
				int rightEnd = (leftStart + 2 * currentSize - 1 < n - 1) ? leftStart + 2 * currentSize - 1 : n - 1;
				int leftSize = mid - leftStart + 1;
				int rightSize = rightEnd - mid;
				int* left = (int*)malloc(leftSize * sizeof(int));
				int* right = (int*)malloc(rightSize * sizeof(int));

				for (int i = 0; i < leftSize; i++) {
					//if (currentSize > 512 * 1 && leftStart > 2095103) printf("########\n");
					left[i] = *(arr + leftStart + i);
				}
				//if (currentSize > 512 * 1 && leftStart > 2095103) printf("here2\n");
				for (int i = 0; i < rightSize; i++) {
					right[i] = *(arr + mid + 1 + i);
				}

				_merge(arr + leftStart, left, leftSize, right, rightSize);
				//printf("after : arr0:%d , arr1:%d , arr2:%d , arr3:%d\n", arr[0], arr[1], arr[2], arr[3]);
				free(left);
				free(right);

				__syncthreads();

				nPart /= 2;

			}
			
	
}

__global__ void mergeSortWithSharedMemmory(int* arr, int n) {
	extern __shared__ int sharedArray[];

	int currentSize, leftStart;

	for (currentSize = 1; currentSize < n; currentSize *= 2) {
		leftStart = blockIdx.x * (2 * currentSize);
		int mid = leftStart + currentSize - 1;
		int rightEnd = (leftStart + 2 * currentSize - 1 < n - 1) ? leftStart + 2 * currentSize - 1 : n - 1;
		int leftSize = mid - leftStart + 1;
		int rightSize = rightEnd - mid;

		int* left = &sharedArray[leftStart];
		int* right = &sharedArray[mid + 1];

		// Copy data to shared memory
		if (threadIdx.x < leftSize)
			left[threadIdx.x] = arr[leftStart + threadIdx.x];
		if (threadIdx.x < rightSize)
			right[threadIdx.x] = arr[mid + 1 + threadIdx.x];

		__syncthreads();

		_merge(arr + leftStart, left, leftSize, right, rightSize);

		__syncthreads();
	}
}

// Helper function checks validation of cuda codes and merge sort as sequential
hipError_t mergeSortHelper(int* arr, size_t size){
	hipError_t cudaStatus;
	int* d_arr;
	double tstart, tend;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
	}

	cudaStatus = hipMalloc((void**)&d_arr, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
	}

	cudaStatus = hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
	}

	// Launch merge sort on the GPU with one thread and one block.
	tstart = get_time();
	mergeSort <<<1, 1 >>> (d_arr, N);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "mergeSort launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipDeviceSynchronize();
	tend = get_time();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mergeSort!\n", cudaStatus);
		fprintf(stderr, "Error is: %s \n",hipGetErrorString(cudaStatus));
	}

	showTime(tstart, tend, (char*)"Sequential Implementation");
	cudaStatus = hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
	}

	hipFree(d_arr);

	return cudaStatus;

}

// Helper function checks validation of cuda codes and merge sort as sequential
hipError_t mergeSortWithThreadsHelper(int* arr, size_t size) {
	hipError_t cudaStatus;
	int* d_arr;
	double tstart, tend;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
	}

	cudaStatus = hipMalloc((void**)&d_arr, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
	}

	cudaStatus = hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
	}

	// Launch merge sort on the GPU with one thread and one block.
	tstart = get_time();
	//mergeSortWithThreads << <1, BLKDIM >> > (d_arr, N);
	//mergeSortWithBlocks << <N, 1 >> > (d_arr, N);
	//mergeSortWithBlocksAndThreads << <(N + BLKDIM -1 )/ (BLKDIM * 2), BLKDIM >> > (d_arr, N);
	//mergeSortWithThreadsAndBlocks_limited << <BLKSIZE , BLKDIM >> > (d_arr, N);
	mergeSortWithThreadsAndBlocks_limited_shared << <BLKSIZE , BLKDIM >> > (d_arr, N);
	//mergeSortWithSharedMemmory << <1, 1, N * sizeof(int) >> > (d_arr, N);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "mergeSortWithThreads launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipDeviceSynchronize();
	tend = get_time();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mergeSortWithThreads!\n", cudaStatus);
		fprintf(stderr, "Error is: %s \n", hipGetErrorString(cudaStatus));
	}

	showTime(tstart, tend, (char*)"Parallel Implementation with Threads");
	cudaStatus = hipMemcpy(arr, d_arr, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
	}

	hipFree(d_arr);

	return cudaStatus;

}


int main() {
	int *arr;
	const size_t size = N * sizeof(int);
	hipError_t cudaStatus;
	arr = (int*)malloc(size);

	printf("Calculation Is Starting ...\n");

	// Sequential Implementation
	printf("\n########Sequential Implementation########\n");
	//initilizeArray(arr);
	//// Host code: demonstrating the array before sorting
	//printf("The orginal array is (%d Items): \n", SHOWITEMS);
	//printArray(arr, N);
	//cudaStatus = mergeSortHelper(arr, size);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "mergeSortHelper failed!\n");
	//	// Todo: another alternative would be exit program after receiving error in any step
	//	//return 1;
	//}
	//printf("The sorted array is: \n");
	//printArray(arr, N);

	// Parallel Implementation with threads and one block
	printf("\n########Parallel Impelementation########\n");
	initilizeArray(arr);
	printf("The orginal array is (%d Items): \n", SHOWITEMS);
	printArray(arr, N);
	cudaStatus = mergeSortWithThreadsHelper(arr, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "mergeSortWithThreadsHelper failed!\n");
		//return 1;
	}
	printf("The sorted array is: \n");
	printArray(arr, N);

	printf("\nCalculation Is Finished!\n");

	free(arr);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

